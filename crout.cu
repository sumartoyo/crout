
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define n 4

__device__
void dekomposisi(double A[][n], double D[][n]) {
    int i, j, k, p, q, stop = 0;
    double sum = 0;

    for (p = 0; p < n; p++) {
        for (j = p; j < n; j++) {
            sum = 0;
            for (k = 0; k < p; k++) {
                sum += D[p][k] * D[k][j];
            }
            D[p][j] = A[p][j] - sum;
        }

        q = p;
        for (i = q + 1; i < n; i++) {
            sum = 0;
            for(k = 0; k < q; k++) {
                sum += D[i][k] * D[k][q];
            }
            if (D[q][q] == 0) {
                printf("U[%d][%d] == 0\n Tidak bisa dibagi 0...\n", q, q);
                stop = 1;
            } else {
                D[i][q] = (A[i][q] - sum) / D[q][q];
            }

            if (stop) {
                break;
            }
        }

        if (stop) {
            break;
        }
    }
}

__device__
void sulih(double D[][n], double b[n], double x[n], double *y) {
    int i, j;
    double sum;

    for (i = 0; i < n; i++) {
        sum = 0;
        for (j = 0; j < i; j++) {
            sum += y[j] * D[i][j];
        }
        y[i] = b[i] - sum;
    }

    for (i = n-1; i >= 0; i--) {
        sum = 0;
        for (j = i+1; j < n; j++) {
            sum += x[j] * D[i][j];
        }
        x[i] = (y[i] - sum) / D[i][i];
    }

    free(y);
}

__device__
void print_LU(double D[][n]) {
    int i, j;
    printf("Dekomposisi\n");

    printf("L =\n");
    for (i = 0; i < n; i++) {
        printf("\t");
        for (j = 0; j < i; j++) {
            printf("%8.4f  ", D[i][j]);
        }
        printf("%8d\n", 1);
    }

    printf("U =\n");
    for (i = 0; i < n; i++) {
        printf("\t");
        for (j = 0; j < i; j++) {
            printf("%8s  ", "");
        }
        for (j = i; j < n; j++) {
            printf("%8.4f  ", D[i][j]);
        }
        printf("\n");
    }
}

__device__
void print_x(double x[n]) {
    int i;
    printf("Solusi\n");

    printf("x =\n");
    for (i = 0; i < n; i++) {
        printf("\t%8.4f\n", x[i]);
    }
}

__global__
void crout(double *y) {
    double A[][n] = {
        {0.31, 0.14, 0.30, 0.27},
        {0.26, 0.32, 0.18, 0.24},
        {0.61, 0.22, 0.20, 0.31},
        {0.40, 0.34, 0.36, 0.17},

        // {0.7071, 0,  1,  0,     0.5,  0,  0,  0,       0},
        // {0,      1,  0,  0,       0, -1,  0,  0,       0},
        // {0,      0, -1,  0,       0,  0,  0,  0,       0},
        // {0,      0,  0,  1,       0,  0,  0,  0, -0.7071},
        // {0.7071, 0,  0, -1, -0.8660,  0,  0,  0,       0},
        // {0,      0,  0,  0,       0,  0,  1,  0,  0.7071},
        // {0,      0,  0,  0,    -0.5,  0, -1,  0,       0},
        // {0,      0,  0,  0,  0.8660,  1,  0, -1,       0},
        // {0,      0,  0,  0,       0,  0,  0,  0,  0.7071},

        // { 0.866,  0,   -0.5,  0,  0,  0},
        // {     0,  1,    0.5,  0,  0,  0},
        // {   0.5,  0,  0.866,  0,  0,  0},
        // {-0.866, -1,      0, -1,  0,  0},
        // {  -0.5,  0,      0,  0, -1,  0},
        // {     0,  0, -0.866,  0,  0, -1},
    };
    double b[n] = {
        1.02,
        1.00,
        1.34,
        1.27,

        // -1000,
        // 0,
        // 0,
        // 0,
        // 0,
        // 500,
        // -500,
        // 0,
        // 0,

        // 0,
        // 0,
        // -1000,
        // 0,
        // 0,
        // 0,
    };

    double D[][n] = {
        {0, 0, 0, 0},
        {0, 0, 0, 0},
        {0, 0, 0, 0},
        {0, 0, 0, 0},
        {0, 0, 0, 0},
        {0, 0, 0, 0},
    };
    double x[n] = {
        0,
        0,
        0,
        0,
    };

    dekomposisi(A, D);
    print_LU(D);
    sulih(D, b, x, y);
    print_x(x);
}

int main(int argc, char *argv[] ) {
    double *y;
    hipMalloc(&y, n*sizeof(double));
    crout<<<1, 1>>>(y);
    hipDeviceSynchronize();
    return 0;
}
