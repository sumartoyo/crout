
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define err 0.000001

__device__
void f(float x, float *y)
{
    // *y = exp(x)-5*pow(x,2); // slide
    *y = ((70 + 1.463/pow(x, 2)) * (x - 0.0394)) - (0.08314 * 215);
}

__device__
void g(float x, float *y)
{
    // *y = exp(x)-10*x; // slide
    *y = 70 - 1.463 + 2*1.463*0.0394;
}

__global__
void newtraph()
{
    float x,xS,fx,gx,fS,gS;
    x = 1;

    printf("%11s %11s %11s %11s\n",
        "x", "f(x)", "f'(x)", "x");
    do
    {
        xS=x;
        f(x, &fx);
        g(x, &gx);
        f(xS, &fS);
        g(xS, &gS);
        x=x-fx/gx;
        printf("%11.6f %11.6f %11.6f %11.6f\n",
            xS,fS,gS,x);
    }
    while(fabs(x-xS)>err);
    printf("Hasil = %.6f\n",x);
}

int main(int argc,char **argv) {
    newtraph<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
