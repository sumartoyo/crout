
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define err 0.00001

__device__
void f(float x, float *y)
{
    // *y = exp(x)-5*pow(x,2); // slide
    *y = (pow(x, 2)*(2.1-0.5*x)/(pow(1-x, 2)*(1.1-0.5*x)))-13.616; // 1.a
    // *y = tan(x) - x + 1; // 1.b
    // *y = 0.5*exp(x/3) - sin(x); // 1.c
}

__global__
void bisection()
{
    float x0,x1,x2,y0,y1,y2;

    // x0 = 0; x1 = 1; // slide
    x0 = -0.1; x1 = 1.1; // 1.a
    // x0 = 0; x1 = 3*M_PI; // 1.b
    // x0 = 0; x1 = 1; // 1.c

    printf("%10s %10s %10s %10s %10s %10s\n",
        "x0", "x1", "f(x0)", "f(x1)", "x2", "f(x2)");
    do
    {
        x2=(x0+x1)/2;
        f(x0, &y0);
        f(x1, &y1);
        f(x2, &y2);
        printf("%10.5f %10.5f %10.5f %10.5f %10.5f %10.5f\n",
            x0,x1,y0,y1,x2,y2);
        if(y0*y2<0) x1=x2;
        else x0=x2;
    }
    while(fabs(x0-x1)>err);

    printf("Hasil = %.5f\n",x2);
}

int main(int argc,char **argv) {
    bisection<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}

