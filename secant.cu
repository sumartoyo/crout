
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define err 0.00001

__device__
void f(float x, float *y)
{
    // *y = exp(x)-5*pow(x,2); // slide
    // *y = (pow(x, 2)*(2.1-0.5*x)/(pow(1-x, 2)*(1.1-0.5*x)))-13.616;
    *y = tan(x) - x + 1; // 1.b
    // *y = 0.5*exp(x/3) - sin(x); // 1.c
}

__global__
void secant()
{
    float x0,x1,x2,xS,y0,y1;

    // x0 = 0.5; x1 = 1; // slide
    // x0 = -0.2; x1 = 1.2; // 1.a
    x0 = 0; x1 = 3*M_PI; // 1.b
    // x0 = 0; x1 = 1; // 1.c

    printf("%10s %10s %10s %10s %10s\n",
        "x0", "x1", "f(x0)", "f(x1)", "x2");
    do
    {
        f(x0, &y0);
        f(x1, &y1);
        xS=x1;
        x2=x1-(y1*(x1-x0)/(y1-y0));
        printf("%10.5f %10.5f %10.5f %10.5f %10.5f\n",
            x0,x1,y0,y1,x2);
        if (fabs(x2-xS)<err) {
            xS=x2;
        } else {
            x0=x1;
            x1=x2;
        }
    }
    while(fabs(x2-xS)>err);

    printf("Hasil = %.5f\n",x2);
}

int main(int argc, char **argv) {
    secant<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}